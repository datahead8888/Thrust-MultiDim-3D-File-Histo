#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudaTimer.h"
#include "windowsCpuTimer.h"

#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <opencv2/opencv.hpp>

#include <Windows.h>

using namespace std;
using namespace cv;

bool loadImage(string fileName, Mat & image)
{
	image = imread(fileName);

	if (image.empty())
	{
		cerr << "Error in loading image" << endl;
		return false;
	}

	cout << "Image dimensions: " << image.cols << " X " << image.rows << endl;

	return true;

	
}

bool generateRandomData(int rows, int cols, int max, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			data[i * cols + j] = rand() % max + 1;

		}
	}

	return true;

}

void printData(int rows, int printWidth, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		cout << setw(printWidth) << data[i] << endl;
	
	}

}

void printData(int rows, int printWidth, thrust::device_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		cout << setw(printWidth) << data[i] << endl;
	
	}

}

void printData(int rows, int cols, int printWidth, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printData(int rows, int cols, int printWidth, thrust::device_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printHistoData(int rows, int cols, int printWidth, thrust::host_vector<int> & multiDimKeys, thrust::host_vector<int> & counts)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << multiDimKeys[i * cols + j];

		}

		cout << setw(printWidth) << "*" << counts[i];

		cout << endl;
	}

}

thrust::host_vector<int> doHistogramGPU(int ROWS, int COLS, int MAX)
{
	
	thrust::host_vector<int> h_data(COLS * ROWS);
	
	generateRandomData(ROWS, COLS, MAX, h_data);
	
	#ifdef IS_LOGGING	
	printData(ROWS, COLS, 5, h_data);
	#endif
	
	thrust::device_vector<int>d_data(h_data.begin(), h_data.end());

	//auto zipFirst = thrust::make_zip_iterator(thrust::make_tuple(d_red_vector.begin(), d_green_vector.begin(), d_blue_vector.begin()));
	//auto zipLast = thrust::make_zip_iterator(thrust::make_tuple(d_red_vector.end(), d_green_vector.end(), d_blue_vector.end()));
	auto zipFirst = thrust::make_zip_iterator(thrust::make_tuple(d_data.begin()));
	auto zipLast = thrust::make_zip_iterator(thrust::make_tuple(d_data.end()));

	
	CudaTimer cudaTimer;
	WindowsCpuTimer cpuTimer;
	
	//Reference: http://stackoverflow.com/questions/1739259/how-to-use-queryperformancecounter
	
	//Timing code start
	

	#ifdef IS_LOGGING
	cout << "Running multidimensional histogram GPU method..." << endl;
	cout << endl;
	#endif

	cudaTimer.startTimer();
	cpuTimer.startTimer();
	
	#ifdef IS_LOGGING
	cout << "Running transform:" << endl;
	#endif

	
	//Phase 1: Find the bins for each of the elements

    thrust::transform(zipFirst, zipLast, zipFirst, zipFirst, BinFinder());

	#ifdef IS_LOGGING
	cout << "Printing bin assignment" << endl;
	printData(ROWS, COLS, 5, d_data);
	#endif
	

	//Phase 2: Convert this effectively multi-dimensional vector into a one dimensional vector
	
	//TO DO: Parallelize this

	//////////////////////////////////////////
	h_data = d_data; //Copy from device_vector back to host_vector, since this code is currently executed on the CPU

	thrust::host_vector<int> h_single_data(ROWS);

	for (int i = 0; i < ROWS; i++)
	{
		h_single_data[i] = 0;
		int factor = 1;
		for (int j = COLS - 1; j >= 0; j--)
		{
			h_single_data[i] += (h_data[i * COLS + j] - 1) * factor;

			factor *= 4;

		}
	}

	#ifdef IS_LOGGING	
	cout << "Printing 1-D representation of data - from CPU" << endl;
	printData(ROWS, 5, h_single_data);
	#endif

	/////////////////////////////////////////////////////////////////////////////////

	thrust::device_vector<int> d_single_data(ROWS);


	thrust::counting_iterator<int> counter;
	auto zipStart = thrust::make_zip_iterator(thrust::make_tuple(d_single_data.begin(), counter));
	auto zipEnd = thrust::make_zip_iterator(thrust::make_tuple(d_single_data.end(), counter + d_data.size()));

	thrust::device_ptr<int> devPtr = &d_data[0];

	thrust::for_each(zipStart, zipEnd, MultiToSingleDim(thrust::raw_pointer_cast(devPtr)));

	//thrust::raw_pointer_cast(devPtr);
	///////////thrust::for_each(zipStart, zipEnd, MultiToSingleDim());

	
	
	/*
	thrust::transform(
		thrust::make_permutation_iterator(h_data.begin(), thrust::make_transform_iterator(counter, Decimator(4))), 
		thrust::make_permutation_iterator(h_data.begin() + ROWS * COLS, thrust::make_transform_iterator(counter, Decimator(4))), 
		d_single_data.begin(), 
		PowerSeries(4));
		*/
	//thrust::make_transform_iterator(counter, Decimator(4));
	//thrust::make_permutation_iterator(h_data.begin(), thrust::make_transform_iterator(counter, Decimator(4)));
	//thrust::make_permutation_iterator(h_data.begin() + ROWS * COLS, thrust::make_transform_iterator(counter, Decimator(4)));

	#ifdef IS_LOGGING	
	cout << "Printing 1-D representation of data - from GPU - Prelim" << endl;
	printData(ROWS, 5, d_single_data);
	#endif

	cout << endl;

	

	//thrust::device_vector<int> d_single_data(h_single_data.begin(), h_single_data.end());

	//auto singleZipFirst = thrust::make_zip_iterator(thrust::make_tuple(d_single_data.begin()));
	//auto singleZipLast = thrust::make_zip_iterator(thrust::make_tuple(d_single_data.end()));

	
	////Step 2: Sort those bin ids
	thrust::sort(d_single_data.begin(), d_single_data.end());

	#ifdef IS_LOGGING	
	cout << "Printing SORTED 1-D representation of data" << endl;
	printData(ROWS, 5, d_single_data);
	#endif

	////Step 3: Use the reduce by key function to get a count of each bin type
	thrust::constant_iterator<int> cit(1);
	thrust::device_vector<int> d_counts(h_single_data.size());  //4 ^ 3

	typedef thrust::device_vector<int>::iterator DVI;

	thrust::pair<DVI, DVI> endPosition = thrust::reduce_by_key(d_single_data.begin(), d_single_data.end(), cit, d_single_data.begin(), d_counts.begin());

	#ifdef IS_LOGGING
	
	cout << "Results after reduce key: " << endl;

	cout << "Keys (the 1-d representation of data): " << endl;

	for (DVI it = d_single_data.begin(); it != endPosition.first; it++)
	{
		cout << setw(4) << *it << " ";
	}
		
	cout << endl << "Counts:" << endl;

	for (DVI it = d_counts.begin(); it != endPosition.second; it++)
	{
		cout << setw(4) << *it << " ";
	}

	cout << endl;
	cout << endl;
	#endif
	
	thrust::host_vector<int> final_data (d_single_data.size() * COLS);

	//Multidimensional representation reconstruction
	int i = 0;
	for (DVI it = d_single_data.begin(); it != endPosition.first; it++, i++)
	{
		int value = *it;

		for (int j = COLS - 1; j >= 0; j--)
		{
			int moddedValue = value % 4 + 1;
			final_data[i * COLS + j] = moddedValue;
			value /= 4;

		}
	}

	#ifdef IS_LOGGING
	printHistoData(i, COLS, 5, final_data, thrust::host_vector<int>(d_counts.begin(), d_counts.end()));
	#endif

	
	

	

	
	
	cudaTimer.stopTimer();
	cpuTimer.stopTimer();

	cout << "GPU time elapsed for GPU method #2: " << cudaTimer.getTimeElapsed() << endl;
	
	

	cout << "CPU time elapsed for GPU method #2: " << cpuTimer.getTimeElapsed() << endl;
	

	return final_data;

	

}

std::vector<int> doHistogramCPU(int ROWS, int COLS, int MAX)
{
	thrust::host_vector<int> h_data(COLS * ROWS);
	
	generateRandomData(ROWS, COLS, MAX, h_data);
	
	#ifdef IS_LOGGING
	cout << "Random data:" << endl;
	printData(ROWS, COLS, 5, h_data);
	#endif
	
	//Reference: http://stackoverflow.com/questions/1739259/how-to-use-queryperformancecounter
	//Timing code start
	WindowsCpuTimer cpuTimer;
	cpuTimer.startTimer();
	
	
	#ifdef IS_LOGGING
	cout << "Running histogram CPU Method..." << endl;
	cout << endl;
	#endif

	//Calculate the number of elements belonging in each bin on the CPU using a for loop
	
	int numElements = 1;
	for (int i = 0; i < COLS; i++)
	{
		numElements *= 4;
	}


	std::vector<int> finalCounts(numElements);

	for (int i = 0; i < finalCounts.size(); i++)
	{
		finalCounts[i] = 0;
	}


	for (int i = 0; i < ROWS; i++)
	{
		int factor = 1;
		int sum = 0;
		for (int j = COLS - 1; j >= 0; j--)
		{
			//sum += (h_data[i * COLS + j] - 1) * factor;
			int value = h_data[i * COLS + j];

			int binValue = 0;

			if (value <= 5)
			{
				binValue = 0;
			}
			else if (value >= 6 && value <= 9)
			{
				binValue = 1;
			}
			else if (value >= 10 && value <= 14)
			{
				binValue = 2;
			}
			else
			{
				binValue = 3;
			}

			sum += binValue * factor;

			factor *= 4;

		}

		finalCounts[sum]++;
	}

	//Timing code end
	cpuTimer.stopTimer();

	#ifdef IS_LOGGING
	cout << "Generated histogram:" << endl;
	printData(finalCounts.size(), 5, thrust::host_vector<int>(finalCounts.begin(), finalCounts.end()));

	cout << endl;
	#endif
	


	cout << "CPU time elapsed for CPU method: " << cpuTimer.getTimeElapsed() << endl;

	return finalCounts;

}

