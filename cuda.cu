#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudaTimer.h"
#include "windowsCpuTimer.h"

#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
//#include <vtkExecutive.h>
//#include <vtkStructuredPointsReader.h>
//#include <vtkAlgorithm.h>

#include <Windows.h>

using namespace std;


bool loadTextFile(FILE *infile, int xSize, int ySize, int zSize, int numvars, int maxVars, thrust::host_vector<float> & h_data, int bufferSize, int & xPos, int & yPos, int & zPos )
{

	WindowsCpuTimer cpuTimer;

	cpuTimer.startTimer();

	
	
	
	//Data from http://sciviscontest.ieeevis.org/2008/data.html
	//fscanf code below is also partially borrowed from those pages

	float currentValue = 0;
	int recordsRead = 0;

	for (int z = zPos; z < zSize; z++)
	{
		for (int y = yPos; y < ySize; y++)
		{
			for (int x = xPos; x < xSize; x++)
			{
				bool hadEOF = false;
				for (int v = 0; v < numvars; v++)
				{

					fscanf(infile, "%f", &currentValue);

					if (feof(infile))
					{
						hadEOF = true;
						break;
					}

					#ifdef PRINT_INPUT
					cout << "x = " << x << " y = " << y << " z = " << z << " v = " << v << endl;
					//cout << "Density: " << density << " Temperature: " << temperature << " ab_H " << ab_H << " ab_HP " << ab_HP << " ab_He " << ab_He << " ab_HeP " << ab_HeP << " ab_HEPP " << ab_HePP << " ab_HM " << ab_HM << " ab_H2 "<< ab_H2 << " ab_H2P " << ab_H2P << endl;
					cout << "Value: " << currentValue << endl;
					#endif

					h_data[recordsRead * numvars + v] = currentValue;

				} //END: for (int v = 0; v < numvars && keepGoing; v++)

				
				//If less variables are requested than are in the file (currently 10), burn through variables until we get to the next record
				if (!hadEOF && numvars < maxVars)
				{
					for (int v = 0; v < maxVars - numvars; v++)
					{
						fscanf(infile, "%f", &currentValue);

						if (feof(infile))
						{
							hadEOF = true;
							break;
						}


					}
				} //END: if (!hadEOF && numvars < maxVars)

				recordsRead++;

				if (recordsRead == bufferSize || hadEOF)
				{
					cpuTimer.stopTimer();

					cout << "File load time: " << cpuTimer.getTimeElapsed() << endl;

					
					//Hacky code to store the proper x, y, and z values to pick up on the for loop next time
					x++;
					
					if (x >= xSize)
					{
						y++;
						x = 0;
					}
					
					if (y >= ySize)
					{
						z++;
						x = 0;
						y = 0;
					}
										

					
					xPos = x; yPos = y; zPos = z;

					if (x >= xSize && y >= ySize && z >= zSize)
					{
						//Would have exited loop if didn't have a file size that is a multiple of the buffer size.  Return false to end the loop in the main function
						return false;
					}
					else
					{
						//More data remains in the file.  Return true to keep that loop in the main function going.
						return true;
					}
				}

			}
		}
	}

	//If records were read, we will return true so that the loop that calls this can do one more iteration.
	//It will then try to call this function again.  We need to set the x, y, and z starting positions so that no records will be read next time.
	xPos = xSize;
	yPos = ySize;
	zPos = zSize;

	if (recordsRead < bufferSize)
	{
		h_data.resize(recordsRead * numvars);
	}

	cpuTimer.stopTimer();
	cout << "File load time: " << cpuTimer.getTimeElapsed() << endl;


	if (recordsRead == 0)
	{
		return false;
	}
	else
	{
		return true;
	}


}

bool generateRandomData(int rows, int cols, int max, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			data[i * cols + j] = rand() % max + 1;

		}
	}

	return true;

}

void printData(int rows, int printWidth, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		cout << setw(printWidth) << data[i] << endl;
	
	}

}

void printDataNoZeroes(int rows, int printWidth, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		if (data[i] != 0)
		{
			cout << "i = " << i << ":" << setw(printWidth) << data[i] << endl;
		}
	
	}

}

void printData(int rows, int printWidth, thrust::device_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		cout << setw(printWidth) << data[i] << endl;
	
	}

}

void printData(int rows, int cols, int printWidth, thrust::host_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printData(int rows, int cols, int printWidth, thrust::device_vector<int> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printData(int rows, int cols, int printWidth, thrust::device_vector<float> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printData(int rows, int cols, int printWidth, thrust::host_vector<float> & data)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << data[i * cols + j];

		}
		cout << endl;
	}

}

void printHistoData(int rows, int cols, int printWidth, thrust::host_vector<int> & multiDimKeys, thrust::host_vector<int> & counts)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			cout << setw(printWidth) << multiDimKeys[i * cols + j];

		}

		cout << setw(printWidth) << "*" << counts[i];

		cout << endl;
	}

}

void doHistogramGPU(int xSize, int ySize, int zSize, int numVars, thrust::host_vector<float> & h_buffer, thrust::host_vector<int> & h_data, thrust::host_vector<int> & h_data2, int numBins, CudaTimer & cudaTimer, WindowsCpuTimer & cpuTimer)
{
	
	thrust::device_vector<float>d_data(h_buffer.begin(), h_buffer.end());
	thrust::device_vector<int>d_bins(h_buffer.size());

	auto zipInFirst = thrust::make_zip_iterator(thrust::make_tuple(d_data.begin()));
	auto zipInLast = thrust::make_zip_iterator(thrust::make_tuple(d_data.end()));
	auto zipOutFirst = thrust::make_zip_iterator(thrust::make_tuple(d_bins.begin()));
	thrust::counting_iterator<int> counter(0);
	
	
	
	//Reference: http://stackoverflow.com/questions/1739259/how-to-use-queryperformancecounter
	
	//Timing code start
	

	#ifdef IS_LOGGING
	cout << "Running multidimensional histogram GPU method..." << endl;
	cout << endl;

	#endif

	cudaTimer.startTimer();
	cpuTimer.startTimer();
	
	#ifdef IS_LOGGING
	cout << "Running transform:" << endl;
	#endif

	
	//Phase 1: Find the bins for each of the elements
	
	float minValues[] = {0, 0, 0, 0, 0, 0, 7.392e-039, 0, 0, 0};
	float maxValues[] = {1001, 19910, 0.7599, 0.7595, 0.24, 0.2397, 0.1623, 1.1e-007, 3.464e-006, 4.447e-008};

	thrust::device_vector<float> d_minValues(minValues, minValues+10);
	thrust::device_vector<float> d_maxValues(maxValues, maxValues+10);

	#ifdef IS_LOGGING
	cout << "Min values vector:" << endl;
	for (int i = 0; i < d_minValues.size(); i++)
	{
		cout << d_minValues[i] << " ";
	}
	cout << endl;

	cout << "Max values vector:" << endl;
	for (int i = 0; i < d_maxValues.size(); i++)
	{
		cout << d_maxValues[i] << " ";
	}
	cout << endl;

	
	#endif

	thrust::device_ptr<float> minDevPtr = &d_minValues[0];
	thrust::device_ptr<float> maxDevPtr = &d_maxValues[0];

    thrust::transform(zipInFirst, zipInLast, counter, zipOutFirst, BinFinder(thrust::raw_pointer_cast(minDevPtr), thrust::raw_pointer_cast(maxDevPtr), numVars, numBins));

	#ifdef IS_LOGGING
	cout << "Printing bin assignment" << endl;
	printData(h_buffer.size() / numVars, numVars, 10, d_bins);
	#endif

	cout << endl;
	

	////Phase 2: Convert this effectively multi-dimensional vector into a one dimensional vector

	thrust::device_vector<int> d_single_data(h_buffer.size() / numVars);

	thrust::constant_iterator<int> colCountIt(numVars);
	//thrust::counting_iterator<int> counter(0);
	auto zipStart = thrust::make_zip_iterator(thrust::make_tuple(counter, colCountIt, d_single_data.begin()));
	auto zipEnd = thrust::make_zip_iterator(thrust::make_tuple(counter + d_single_data.size(), colCountIt + d_single_data.size(), d_single_data.end()));

	thrust::device_ptr<int> devPtr = &d_bins[0];

	thrust::for_each(zipStart, zipEnd, MultiToSingleDim(thrust::raw_pointer_cast(devPtr), numBins));

	#ifdef IS_LOGGING	
	cout << "Printing 1-D representation of data - from GPU - Prelim" << endl;
	printData(h_buffer.size() / numVars, 10, d_single_data);
	#endif

	//cout << endl;
	//
	//////Step 2: Sort those bin ids
	thrust::sort(d_single_data.begin(), d_single_data.end());

	#ifdef IS_LOGGING	
	cout << "Printing SORTED 1-D representation of data" << endl;
	printData(h_buffer.size() / numVars, 10, d_single_data);
	#endif

	//////Step 3: Use the reduce by key function to get a count of each bin type
	thrust::constant_iterator<int> cit(1);
	thrust::device_vector<int> d_counts(d_single_data.size());  //4 ^ 3

	//typedef thrust::device_vector<int>::iterator DVI;

	thrust::pair<DVI, DVI> endPosition = thrust::reduce_by_key(d_single_data.begin(), d_single_data.end(), cit, d_single_data.begin(), d_counts.begin());

	int numElements = endPosition.first - d_single_data.begin();
	
	#ifdef IS_LOGGING

	cout << "Number of elements from reduce key: " << numElements << endl;
	
	cout << "Results after reduce key: " << endl;

	cout << "Keys (the 1-d representation of data): " << endl;

	for (DVI it = d_single_data.begin(); it != endPosition.first; it++)
	{
		cout << setw(4) << *it << " ";
	}
		
	cout << endl << "Counts:" << endl;

	for (DVI it = d_counts.begin(); it != endPosition.second; it++)
	{
		cout << setw(4) << *it << " ";
	}
	
	cout << endl;
	cout << endl;
	#endif
	
	h_data.insert(h_data.begin(), d_single_data.begin(), endPosition.first);
	h_data2.insert(h_data2.begin(), d_counts.begin(), endPosition.second);
	
	
	
	cudaTimer.stopTimer();
	cpuTimer.stopTimer();

	/*
	#ifdef IS_LOGGING
	cout << "Final multidimensional representation from GPU" << endl;
	printHistoData(h_buffer.size() / numVars, numVars, 10, thrust::host_vector<int>(d_final_data.begin(), d_final_data.end()), thrust::host_vector<int>(d_counts.begin(), d_counts.end()));
	#endif
	*/

	cout << "GPU time elapsed for GPU method: " << cudaTimer.getTimeElapsed() << endl;

	cout << "CPU time elapsed for GPU method: " << cpuTimer.getTimeElapsed() << endl;
	
	


	

}

//h_data - the keys
//h_data2 - the counts
void histogramMapReduceGPU(thrust::host_vector<int> & h_data, thrust::host_vector<int> & h_data2, thrust::pair<DVI, DVI> & endPosition, int numVars, int numBins, CudaTimer & cudaTimer, WindowsCpuTimer & cpuTimer)
{
	cudaTimer.startTimer();
	cpuTimer.startTimer();
	
	thrust::device_vector<int> d_data(h_data.begin(), h_data.end());
	thrust::device_vector<int> d_data2(h_data2.begin(), h_data2.end());

	
	thrust::sort_by_key(d_data.begin(), d_data.end(), d_data2.begin());

	endPosition = thrust::reduce_by_key(d_data.begin(), d_data.end(), d_data2.begin(), d_data.begin(), d_data2.begin());

	#ifdef IS_LOGGING

	cout << "Did final map reduce..." << endl;
	cout << "GPU Keys:" << endl;                               //The new "d_single_data"

	for (DVI it = d_data.begin(); it != endPosition.first; it++)
	{
		cout << setw(4) << *it << " ";
	}
		
	cout << endl << "Counts:" << endl;

	cout << "GPU Counts:" << endl;

	for (DVI it = d_data2.begin(); it != endPosition.second; it++)
	{
		cout << setw(4) << *it << " ";
	}

	cout << endl;
	#endif
	
	int d_data_size = endPosition.first - d_data.begin();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	////Multidimensional representation construction - GPU...
	thrust::counting_iterator<int> counter(0);
	thrust::constant_iterator<int> colCountIt(numVars);
	
	auto zipStart = thrust::make_zip_iterator(thrust::make_tuple(counter, colCountIt, d_data.begin()));
	auto zipEnd = thrust::make_zip_iterator(thrust::make_tuple(counter + d_data_size, colCountIt + d_data_size, endPosition.first));


	thrust::device_vector<int> d_final_data (d_data_size * numVars);
	thrust::device_ptr<int> devPtr = &d_final_data[0];
	
	////Note: We can use the same zipStart and zipEnd iterators as before; we just use a different kernel and a different raw data pointer
	thrust::for_each(zipStart, zipEnd, SingleToMultiDim(thrust::raw_pointer_cast(devPtr), numBins));

	//WIP Section below
	h_data.clear();
	h_data2.clear();

	h_data.insert(h_data.end(), d_final_data.begin(), d_final_data.end());
	h_data2.insert(h_data2.end(), d_data2.begin(), endPosition.second);

	cudaTimer.stopTimer();
	cpuTimer.stopTimer();

	cout << "GPU time elapsed for GPU map reduce: " << cudaTimer.getTimeElapsed() << endl;

	cout << "CPU time elapsed for GPU map reduce: " << cpuTimer.getTimeElapsed() << endl;
}

std::vector<int> doHistogramCPU(int xSize, int ySize, int zSize, int numVars, thrust::host_vector<float> & h_data)
{		
	//Reference: http://stackoverflow.com/questions/1739259/how-to-use-queryperformancecounter
	//Timing code start

	int rows = xSize * ySize * zSize;

	WindowsCpuTimer cpuTimer;
	cpuTimer.startTimer();

	float minValues[] = {0, 0, 0, 0, 0, 0, 7.392e-039, 0, 0, 0};
	float maxValues[] = {1001, 19910, 0.7599, 0.7595, 0.24, 0.2397, 0.1623, 1.1e-007, 3.464e-006, 4.447e-008};
	
	
	#ifdef IS_LOGGING
	cout << "Running histogram CPU Method..." << endl;
	cout << endl;
	#endif

	//Calculate the number of elements belonging in each bin on the CPU using a for loop
	
	int numElements = 1;
	for (int i = 0; i < numVars; i++)
	{
		numElements *= 4; //numBins!
	}


	std::vector<int> finalCounts(numElements);

	for (int i = 0; i < finalCounts.size(); i++)
	{
		finalCounts[i] = 0;
	}


	for (int i = 0; i < rows; i++)
	{
		int factor = 1;
		int sum = 0;
		for (int j = numVars - 1; j >= 0; j--)
		{
			float value = h_data[i * numVars + j];

			float min = minValues[j];
			float max = maxValues[j];

			float percentage = (value - min) / float(max - min);


			int binValue = percentage * 4;

			if (binValue == 4) //numBins!
			{
				binValue--;
			}

			sum += binValue * factor;

			factor *= 4;

		}

		finalCounts[sum]++;
	}

	//Timing code end
	cpuTimer.stopTimer();

	#ifdef IS_LOGGING
	cout << "Generated histogram:" << endl;
	//printData(finalCounts.size(), 10, thrust::host_vector<int>(finalCounts.begin(), finalCounts.end()));

	printDataNoZeroes(finalCounts.size(), 10, thrust::host_vector<int>(finalCounts.begin(), finalCounts.end()));


	cout << endl;
	#endif
	


	cout << "CPU time elapsed for CPU method: " << cpuTimer.getTimeElapsed() << endl;

	return finalCounts;

}

