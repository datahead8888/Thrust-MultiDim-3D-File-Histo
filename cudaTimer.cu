#include "cudaTimer.h"

CudaTimer::CudaTimer()
{
	reinit();
}

void CudaTimer::reinit()
{
	timeElapsed = 0.0f;
	
}

void CudaTimer::startTimer()
{
	(hipEventCreate(&startEvent));
	(hipEventCreate(&stopEvent));
	hipEventRecord(startEvent, 0);
	
}

void CudaTimer::stopTimer()
{
	float stopWatchTime = 0.0f;
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&stopWatchTime, startEvent, stopEvent);

	timeElapsed += stopWatchTime;
}

int CudaTimer::getTimeElapsed()
{
	return timeElapsed;
}
